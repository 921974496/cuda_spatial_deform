#include "utils.cuh"

extern "C" {

Handle* init_2D_handle(size_t y, size_t x){
    Handle *ret = new Handle();
    ret->set_2D(y, x);
    return ret;
}

Handle* init_3D_handle(size_t z, size_t y, size_t x){
    Handle *ret = new Handle();
    ret->set_3D(z, y, x);
    return ret;
}

void linear_interpolate(Handle* cuda_handle, 
                        float* output, 
                        float* input, 
                        int do_reset){
    cuda_handle->copy_input(input);
    cuda_handle->interpolate_linear();
    cuda_handle->copy_output(output);
    if(do_reset)
        cuda_handle->reset();
}

void check_coords(Handle* cuda_handle, float* coords){
    cuda_handle->check_coords(coords);
}

void cu_scale(Handle* cuda_handle, float scale){
    assert(scale > 0.0 && scale < 1.0);
    cuda_handle->scale(scale);
}

void cu_flip(Handle* cuda_handle, int do_x, int do_y, int do_z){
    cuda_handle->flip(do_x, do_y, do_z);
}

void cu_translate(Handle* cuda_handle, float seg_x, float seg_y, float seg_z){
    cuda_handle->translate(seg_x, seg_y, seg_z);
}

void endding_flag(Handle* cuda_handle){
    cuda_handle->recenter();
}

} // extern "C"
