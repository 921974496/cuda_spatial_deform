#include "utils.cuh"

extern "C" {

Handle* init_2D_handle(size_t y, size_t x){
    Handle *ret = new Handle();
    ret->set_2D(y, x);
    return ret;
}

Handle* init_3D_handle(size_t z, size_t y, size_t x){
    Handle *ret = new Handle();
    ret->set_3D(z, y, x);
    return ret;
}

void test(Handle* cuda_handle, float* output, float* input){
    cuda_handle->copy_input(input);
    cuda_handle->do_nothing();
    cuda_handle->copy_output(output);
}

void check_coords(Handle* cuda_handle, float* coords){
    cuda_handle->check_coords(coords);
}

void cu_scale(Handle* cuda_handle, float scale){
    assert(scale > 0.0 && scale < 1.0);
    cuda_handle->scale(scale);
}

} // extern "C"
