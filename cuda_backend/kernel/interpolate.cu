#include "hip/hip_runtime.h"
#include "interpolate.cuh"
#include <math.h>

__global__ void linear_interplate_2D(float* coords, 
                                    float* img, 
                                    float* output, 
                                    size_t dim_y,
                                    size_t dim_x){
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < dim_x * dim_y){
        // recenter
        coords[index] += (float)dim_y/2.0;
        coords[index + dim_x*dim_y] += (float)dim_x/2.0;

        float coords_y = coords[index];
        float coords_x = coords[index + dim_x*dim_y];
        int index_y = (int)floorf(coords_y);
        int index_x = (int)floorf(coords_x);
        float gap_y = coords_y - index_y;
        float gap_x = coords_x - index_x;

        if (index_y < 0 || index_y >= dim_y -1 ||
            index_x < 0 || index_x >= dim_x -1 ){
            output[index] = 0;
        }
        else{
            output[index] = gap_x * gap_y * img[dim_x * (index_y + 1) + index_x + 1] +
                            (1 - gap_x) * gap_y * img[dim_x * (index_y + 1) + index_x] +
                            gap_x * (1 - gap_y) * img[dim_x * (index_y) + index_x + 1] +
                            (1 - gap_x) * (1 - gap_y) * img[dim_x * index_y + index_x];
        }
    }
    __syncthreads();
}

__global__ void linear_interplate_3D(float* coords, 
                                    float* img, 
                                    float* output,
                                    size_t dim_z,
                                    size_t dim_y,
                                    size_t dim_x){
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y * dim_z;
    size_t dim_xy = dim_x * dim_y;

    if(index < total){
        // recenter
        coords[index] += (float)dim_z/2.0;
        coords[index + total] += (float)dim_y/2.0;
        coords[index + 2 * total] += (float)dim_x/2.0;
   
        float coords_z = coords[index];
        float coords_y = coords[index + total];
        float coords_x = coords[index + 2 * total];
        int index_z = (int)floorf(coords_z);
        int index_y = (int)floorf(coords_y);
        int index_x = (int)floorf(coords_x);
        float gap_z = coords_z - index_z;
        float gap_y = coords_y - index_y;
        float gap_x = coords_x - index_x;

        if (index_y < 0 || index_y >= dim_y - 1 ||
            index_x < 0 || index_x >= dim_x - 1 || 
            index_z < 0 || index_z >= dim_z - 1 ){
            output[index] = 0.0;
        }
        else{
            output[index] =  gap_x * gap_y * gap_z * img[dim_xy * (index_z + 1) + dim_x * (index_y + 1) + index_x + 1] +
                            (1 - gap_x) * gap_y * gap_z * img[dim_xy * (index_z + 1) + dim_x * (index_y + 1) + index_x] +
                            gap_x * (1 - gap_y) * gap_z * img[dim_xy * (index_z + 1) + dim_x * (index_y) + index_x + 1] +
                            (1 - gap_x) * (1 - gap_y) * gap_z * img[dim_xy * (index_z + 1) + dim_x * index_y + index_x] +
                            gap_x * gap_y * (1 - gap_z) * img[dim_xy * index_z + dim_x * (index_y + 1) + index_x + 1] +
                            (1 - gap_x) * gap_y * (1 - gap_z) * img[dim_xy * index_z + dim_x * (index_y + 1) + index_x] +
                            gap_x * (1 - gap_y) * (1 - gap_z) * img[dim_xy * index_z + dim_x * (index_y) + index_x + 1] +
                            (1 - gap_x) * (1 - gap_y) * (1 - gap_z) * img[dim_xy * index_z + dim_x * index_y + index_x];
        }
    }
    __syncthreads();
}
