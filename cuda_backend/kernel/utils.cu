#include "hip/hip_runtime.h"
#include "utils.cuh"

#include "ops_copy.cuh"
#include "spatial_deform.cuh"
#include "interpolate.cuh"

__global__ void set_coords_2D(float* coords, size_t y, size_t x){
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t id_x = index % x;
    size_t id_y = index / x;
    if(index < x * y){
        coords[id_x + id_y * x] = id_y - (float)y/2.0;
        coords[id_x + id_y * x + x*y] = id_x - (float)x/2.0;
    }
    __syncthreads();
}

__global__ void set_coords_3D(float* coords, size_t z, size_t y, size_t x){
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t id_x = index % x;
    size_t id_y = (index / x) % y;
    size_t id_z = index / (x * y);
    if(index < x * y * z){
        coords[index] = id_z - (float)z/2.0;
        coords[index + x * y * z] = id_y - (float)y/2.0;
        coords[index + 2 * x * y * z] = id_x -(float)x/2.0;
    }
    __syncthreads();
}

void Handle::set_2D(size_t y, size_t x){
    is_3D = false;
    dim_x = x;
    dim_y = y;
    total_size = dim_x * dim_y;
    coords_size = total_size * 2;

    std::cout<<"Malloc for 2D image ----------\n"
             <<" dim_x : "<<dim_x
             <<" dim_y : "<<dim_y
             <<" total : "<<total_size<<std::endl;
    std::cout<<"Malloc "<< 6 * total_size * sizeof(float)/1024/1024
             << "MB"<<std::endl;

    checkCudaErrors(hipMalloc((void **)&img,
                            total_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&output,
                            total_size * sizeof(float)));
    checkCudaErrors(hipHostMalloc((void **)&pin_img,
                            total_size * sizeof(float)));
    checkCudaErrors(hipHostMalloc((void **)&pin_output,
                            total_size * sizeof(float)));
   
    checkCudaErrors(hipMalloc((void **)&random,
                            coords_size * sizeof(float)));     

    checkCudaErrors(hipMalloc((void **)&coords,
                        coords_size * sizeof(float)));    
    checkCudaErrors(hipHostMalloc((void **)&pin_coords,
                        coords_size * sizeof(float)));
    
    dim3 threads(min(total_size, (long)512), 1, 1);
    dim3 blocks(total_size/512 + 1, 1, 1);
    set_coords_2D<<<blocks, threads, 0, stream>>>(coords, dim_y, dim_x);
    checkCudaErrors(hipStreamSynchronize(stream));
}

void Handle::set_3D(size_t z, size_t y, size_t x){
    is_3D = true;
    dim_x = x;
    dim_y = y;
    dim_z = z;
    total_size = dim_x * dim_y * dim_z;
    coords_size = total_size * 3;

    std::cout<<"Malloc for 3D image ----------\n"
             <<" dim_x : "<<dim_x
             <<" dim_y : "<<dim_y
             <<" dim_z : "<<dim_z
             <<" total : "<<total_size<<std::endl;

    std::cout<<"Malloc "<< 8 * total_size * sizeof(float)/1024/1024
             << "MB"<<std::endl;

    checkCudaErrors(hipMalloc((void **)&gpu_rot_matrix, 9 * sizeof(float)));

    checkCudaErrors(hipMalloc((void **)&img,
                            total_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&output,
                            total_size * sizeof(float)));
    checkCudaErrors(hipHostMalloc((void **)&pin_img,
                            total_size * sizeof(float)));
    checkCudaErrors(hipHostMalloc((void **)&pin_output,
                            total_size * sizeof(float)));

    checkCudaErrors(hipMalloc((void **)&random,
                            coords_size * sizeof(float)));      

    checkCudaErrors(hipMalloc((void **)&coords,
                        coords_size * sizeof(float)));
    checkCudaErrors(hipHostMalloc((void **)&pin_coords,
                        coords_size * sizeof(float)));
 
    dim3 threads(min(total_size, (long)512), 1, 1);
    dim3 blocks(total_size/512 + 1, 1, 1);
    set_coords_3D<<<blocks, threads, 0, stream>>>(coords, dim_z, dim_y, dim_x);
    checkCudaErrors(hipStreamSynchronize(stream));
}

void Handle::scale(float scale){
    assert(scale <= 1.0 && scale > 0.0);
    dim3 threads(min(coords_size, (long)512), 1, 1);
    dim3 blocks(coords_size/512 + 1, 1, 1);
    device_apply_scale<<<blocks, threads, 0, stream>>>(coords, scale, coords_size);
}

void Handle::flip(int do_x, int do_y, int do_z){
    if(is_3D){
        dim3 threads(min(total_size, (long)512), 1, 1);
        dim3 blocks(total_size/512 + 1, 1, 1);
        flip_3D<<<blocks, threads, 0, stream>>>(coords, dim_z, dim_y, dim_x,
                                                do_z, do_y, do_x);
        checkCudaErrors(hipStreamSynchronize(stream));
    }
    else{
        dim3 threads(min(total_size, (long)512), 1, 1);
        dim3 blocks(total_size/512 + 1, 1, 1);
        flip_2D<<<blocks, threads, 0, stream>>>(coords, dim_y, dim_x, do_y, do_x);
        checkCudaErrors(hipStreamSynchronize(stream));
    }
}

void Handle::host_rotate_2D(float angle){
    float cos_angle = cos(angle);
    float sin_angle = sin(angle);
    dim3 threads(min(total_size, (long)512), 1, 1);
    dim3 blocks(total_size/512 + 1, 1, 1);
    rotate_2D<<<blocks, threads, 0, stream>>>(coords, dim_y, dim_x, cos_angle, sin_angle);
    checkCudaErrors(hipStreamSynchronize(stream));    
}

void Handle::host_rotate_3D(float* rot_matrix){
    checkCudaErrors(hipMemcpyAsync(gpu_rot_matrix, 
                                    rot_matrix, 
                                    9 * sizeof(float),
                                    hipMemcpyHostToDevice, 
                                    stream));
    dim3 threads(min(total_size, (long)512), 1, 1);
    dim3 blocks(total_size/512 + 1, 1, 1);
    rotate_3D<<<blocks, threads, 0, stream>>>(coords, dim_z, dim_y, dim_x, gpu_rot_matrix);
    checkCudaErrors(hipStreamSynchronize(stream));    
}

void Handle::elastic(float sigma, float alpha, float truncate,
                                int mode_type, float c_val){
    // generate random offset by coords.size
    checkCudaErrors(hiprandGenerateUniform(gen, random, coords_size));
    // make the radius of the filter equal to truncate standard deviations
    int lw = int(sigma * truncate + 0.5);
    float sigma2 = sigma * sigma;
    // generate kernel
    float total = 0;
    for(int i = -lw; i < lw + 1; i++){
        kernel_pin[i + lw] = exp(i * i * -0.5 / sigma2);
        total += kernel_pin[i + lw];
    }
    for(int i = -lw; i < lw + 1; i++){
        kernel_pin[i + lw] = kernel_pin[i + lw] / total;
    }

    // Copy kernel
    checkCudaErrors(hipMemcpyAsync(kernel, 
                                    kernel_pin, 
                                    (2 * lw + 1) * sizeof(float),
                                    hipMemcpyHostToDevice,
                                    stream));
    checkCudaErrors(hipStreamSynchronize(stream));
    
    if(is_3D){
        dim3 threads(min(coords_size, (long)512), 1, 1);
        dim3 blocks(coords_size/512 + 1, 1, 1);
        scale_random<<<blocks, threads, 0, stream>>>(random, coords_size);

        gussain_filter_x<<<blocks, threads, 0, stream>>>(random, kernel, lw, dim_z,
                                                    dim_y, dim_x, mode_type, c_val);
        gussain_filter_y<<<blocks, threads, 0, stream>>>(random, kernel, lw, dim_z,
                                                    dim_y, dim_x, mode_type, c_val);
        gussain_filter_z<<<blocks, threads, 0, stream>>>(random, kernel, lw, dim_z,
                                                    dim_y, dim_x, mode_type, c_val);

        plus_offsets<<<blocks, threads, 0, stream>>>(coords, random, coords_size, alpha);
        checkCudaErrors(hipStreamSynchronize(stream));
    }
    else{
        dim3 threads(min(coords_size, (long)512), 1, 1);
        dim3 blocks(coords_size/512 + 1, 1, 1);
        scale_random<<<blocks, threads, 0, stream>>>(random, coords_size);

        gussain_filter_x<<<blocks, threads, 0, stream>>>(random, kernel, lw, 1,
                                                    dim_y, dim_x, mode_type, c_val);
        gussain_filter_y<<<blocks, threads, 0, stream>>>(random, kernel, lw, 1,
                                                    dim_y, dim_x, mode_type, c_val);

        plus_offsets<<<blocks, threads, 0, stream>>>(coords, random, coords_size, alpha);
        checkCudaErrors(hipStreamSynchronize(stream));
    }
}

void Handle::translate(float seg_x, float seg_y, float seg_z){
    if(is_3D){
        dim3 threads(min(total_size, (long)512), 1, 1);
        dim3 blocks(total_size/512 + 1, 1, 1);
        translate_3D<<<blocks, threads, 0, stream>>>(coords, dim_z, dim_y, dim_x,
                                                seg_z, seg_y, seg_x);
        checkCudaErrors(hipStreamSynchronize(stream));
    }
    else{
        dim3 threads(min(total_size, (long)512), 1, 1);
        dim3 blocks(total_size/512 + 1, 1, 1);
        translate_2D<<<blocks, threads, 0, stream>>>(coords, dim_y, dim_x, seg_y, seg_x);
        checkCudaErrors(hipStreamSynchronize(stream));
    }    
}

void Handle::copy_input(float* input){
    memcpy(pin_img, input, total_size * sizeof(float));
    checkCudaErrors(hipMemcpyAsync(img, pin_img, total_size * sizeof(float),
                            hipMemcpyHostToDevice, stream));
}

void Handle::do_nothing(){
    only_copy(output, img, total_size);
}

void Handle::copy_output(float* ret){
    checkCudaErrors(hipMemcpyAsync(pin_output, output, total_size * sizeof(float),
                            hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamSynchronize(stream));
    memcpy(ret, pin_output, total_size * sizeof(float));
}

void Handle::check_coords(float* output){
    checkCudaErrors(hipMemcpyAsync(pin_coords, coords, coords_size * sizeof(float),
                        hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamSynchronize(stream));
    memcpy(output, pin_coords, coords_size * sizeof(float));       
}

void Handle::interpolate(int order){
    dim3 threads(min(total_size, (long)512), 1, 1);
    dim3 blocks(total_size/512 + 1, 1, 1);

    if(is_3D){
        interplate_3D<<<blocks, threads, 0, stream>>>(coords, img, output, order,
                                                            dim_z, dim_y, dim_x, mode_type, c_val);
    }
    else{
        interplate_2D<<<blocks, threads, 0, stream>>>(coords, img, output, order,
                                                             dim_y, dim_x, mode_type, c_val);
    }
}

void Handle::reset(){
    if(is_3D){
        dim3 threads(min(total_size, (long)512), 1, 1);
        dim3 blocks(total_size/512 + 1, 1, 1);
        set_coords_3D<<<blocks, threads, 0, stream>>>(coords, dim_z, dim_y, dim_x);
        checkCudaErrors(hipStreamSynchronize(stream));
    }
    else{
        dim3 threads(min(total_size, (long)512), 1, 1);
        dim3 blocks(total_size/512 + 1, 1, 1);
        set_coords_2D<<<blocks, threads, 0, stream>>>(coords, dim_y, dim_x);
        checkCudaErrors(hipStreamSynchronize(stream));
    }
}

void Handle::recenter(){
    if(is_3D){
        dim3 threads(min(total_size, (long)512), 1, 1);
        dim3 blocks(total_size/512 + 1, 1, 1);
        recenter_3D<<<blocks, threads, 0, stream>>>(coords, dim_z, dim_y, dim_x);
        checkCudaErrors(hipStreamSynchronize(stream));
    }
    else{
        dim3 threads(min(total_size, (long)512), 1, 1);
        dim3 blocks(total_size/512 + 1, 1, 1);
        recenter_2D<<<blocks, threads, 0, stream>>>(coords, dim_y, dim_x);
        checkCudaErrors(hipStreamSynchronize(stream));
    }
}
